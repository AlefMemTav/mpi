#include <stdio.h>
#include <hip/hip_runtime.h>

// nvcc -o cuda_pi cuda_pi.cu
// ./cuda_pi

#define NUM_STEPS 8000000
#define THREADS_PER_BLOCK 256

__global__ void calculate_pi(double step, double* partial_sums, int num_steps) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    double x;
    double sum = 0.0;

    for (int i = idx; i < num_steps; i += stride) {
        x = (i + 0.5) * step;
        sum += 4.0 / (1.0 + x * x);
    }

    partial_sums[idx] = sum;
}

int main() {
    double step = 1.0 / (double)NUM_STEPS;
    double *d_partial_sums, *h_partial_sums;
    double pi = 0.0;

    int num_threads = THREADS_PER_BLOCK;
    int num_blocks = (NUM_STEPS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    size_t partial_sums_size = num_blocks * THREADS_PER_BLOCK * sizeof(double);

    // Alocar memória na CPU e GPU
    h_partial_sums = (double*)malloc(partial_sums_size);
    hipMalloc(&d_partial_sums, partial_sums_size);

    // Lançar o kernel
    calculate_pi<<<num_blocks, num_threads>>>(step, d_partial_sums, NUM_STEPS);

    // Copiar resultados parciais de volta para a CPU
    hipMemcpy(h_partial_sums, d_partial_sums, partial_sums_size, hipMemcpyDeviceToHost);

    // Somar os resultados parciais
    for (int i = 0; i < num_blocks * THREADS_PER_BLOCK; i++) {
        pi += h_partial_sums[i];
    }

    pi *= step;

    // Limpar memória
    free(h_partial_sums);
    hipFree(d_partial_sums);

    printf("Pi = %.15f\n", pi);
    return 0;
}
